#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <THC/THCGeneral.h>
#include <THC/THCDeviceUtils.cuh>

#include <ATen/ATen.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <hip/hip_runtime_api.h>


#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/div_rtn.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include "../cuda/DeviceSqrt.cuh"
#include "../cuda/LaunchUtils.h"
#include <c10/macros/Macros.h>

#define THREAD_1 256
#define THREAD_2 768
#define _stringfy(x) #x
#define STRINGFY(x) _stringfy(x)

namespace at {
namespace native {

template <typename scalar_t, int64_t dim, template <typename U> class PtrTraits = DefaultPtrTraits, typename index_t = int64_t>
static PackedTensorAccessor<scalar_t, dim, PtrTraits, index_t> packed_accessor_or_dummy(const Tensor& t) {
  if (! t.defined()) {
    const std::vector<index_t> zeros(dim);
    return PackedTensorAccessor<scalar_t, dim, PtrTraits, index_t>(nullptr, zeros.data(), zeros.data());
  }
  return t.packed_accessor<scalar_t, dim, PtrTraits, index_t>();
}


using namespace at::cuda;
using namespace at::cuda::detail;

#if defined(__HIP_PLATFORM_HCC__)
constexpr int WARP_SIZE = 64;
#else
constexpr int WARP_SIZE = 32;
#endif

// The maximum number of threads in a block
#if defined(__HIP_PLATFORM_HCC__)
constexpr int MAX_BLOCK_SIZE = 256;
#else
constexpr int MAX_BLOCK_SIZE = 512;
#endif

// Number of threads in a block given an input size up to MAX_BLOCK_SIZE
static int getNumThreads(int nElem) {
#if defined(__HIP_PLATFORM_HCC__)
  int threadSizes[5] = { 16, 32, 64, 128, MAX_BLOCK_SIZE };
#else
  int threadSizes[5] = { 32, 64, 128, 256, MAX_BLOCK_SIZE };
#endif
  for (int i = 0; i != 5; ++i) {
    if (nElem <= threadSizes[i]) {
      return threadSizes[i];
    }
  }
  return MAX_BLOCK_SIZE;
}

// Returns the index of the most significant 1 bit in `val`.
__device__ __forceinline__ int getMSB(int val) {
  return 31 - __clz(val);
}

template <typename scalar_t, typename accscalar_t>
struct Float2 {
  accscalar_t v1, v2;
  __device__ Float2() {}
  __device__ Float2(scalar_t v1, scalar_t v2) : v1(static_cast<accscalar_t>(v1)), v2(static_cast<accscalar_t>(v2)) {}
  __device__ Float2(int v) : v1(static_cast<accscalar_t>(v)), v2(static_cast<accscalar_t>(v)) {}
  __device__ Float2& operator+=(const Float2& a) {
    v1 += a.v1;
    v2 += a.v2;
    return *this;
  }
};

template <typename scalar_t, typename accscalar_t, typename PTA>
struct SumOp {
  __device__ SumOp(const PTA& t) : tensor(t) {}
  __device__ __forceinline__ accscalar_t operator()(int batch, int plane, int n) {
    return static_cast<accscalar_t>(tensor[batch][plane][n]);
  }
  const PTA& tensor;
};

template <typename scalar_t, typename accscalar_t, typename PTA>
struct VarOp {
  __device__ VarOp(accscalar_t m, const PTA& t) : mean(m), tensor(t) {}
  __device__ __forceinline__ accscalar_t operator()(int batch, int plane, int n) {
    accscalar_t val = tensor[batch][plane][n];
    return (val - mean) * (val - mean);
  }
  const accscalar_t mean;
  const PTA& tensor;
};

template <typename scalar_t, typename accscalar_t, typename PTA>
struct GradOp {
  __device__ GradOp(accscalar_t m, const PTA& i, const PTA& g)
    : mean(m), input(i), grad_output(g) {}
  __device__ __forceinline__ Float2<scalar_t, accscalar_t> operator()(int batch, int plane, int n) {
    accscalar_t g = grad_output[batch][plane][n];
    accscalar_t c = static_cast<accscalar_t>(input[batch][plane][n]) - mean;
    return Float2<scalar_t, accscalar_t>(g, g * c);
  }
  const accscalar_t mean;
  const PTA& input;
  const PTA& grad_output;
};

// Sum across all threads within a warp
template <typename T>
static __device__ __forceinline__ T warpSum(T val) {
  for (int i = 0; i < getMSB(WARP_SIZE); ++i) {
    val += WARP_SHFL_XOR(val, 1 << i, WARP_SIZE);
  }
  return val;
}

template <typename scalar_t, typename accscalar_t>
static __device__ __forceinline__ Float2<scalar_t, accscalar_t> warpSum(Float2<scalar_t, accscalar_t> value) {
  value.v1 = warpSum(value.v1);
  value.v2 = warpSum(value.v2);
  return value;
}

// Sum across (batch, x/y/z) applying Op() pointwise
// this works by first having each thread sum it's part
// of the data. Then there is a double-shuffeling reduction.
// First each warp (of WARP_SIZE threads) uses warpSum to reduce its
// data to the "warp leader", who writes its value into shared memory.
// Then a single warp reads the remaining (at most WARP_SIZE) items
// and reduces them using another warpSum.
// The implicit assumption is that there are no more
// than WARP_SIZE**2 threads.
template<typename scalar_t, typename Op, typename PTA>
__device__ scalar_t reduce(Op op, PTA tensor, int plane) {
  // first the reductions each thread does separately
  scalar_t sum = static_cast<scalar_t>(0);
  for (int batch = threadIdx.y; batch < tensor.size(0); batch += blockDim.y) {
    for (int x = threadIdx.x; x < tensor.size(2); x += blockDim.x) {
      sum += op(batch, plane, x);
    }
  }

  // first warpSum to get one value per thread to
  // one value per warp
  sum = warpSum(sum);

  // this writes each warps  item into shared memory
  // there are at most WARP_SIZE items left because
  // there are at most WARP_SIZE**2 threads at the beginning
  __shared__ scalar_t shared[WARP_SIZE];
  __syncthreads();
  int tid = threadIdx.x + threadIdx.y * blockDim.x;
  if (tid % WARP_SIZE == 0) {
    shared[tid / WARP_SIZE] = sum;
  }
  if (tid >= blockDim.x * blockDim.y / WARP_SIZE && tid < WARP_SIZE) {
    // zero out the other entries in shared
    shared[tid] = (scalar_t)0;
  }
  __syncthreads();
  // now have a second warpSum to reduce the intermediate values
  // from shared memory to a single number. The very first
  // thread writes it to shared memory.

  if (tid / WARP_SIZE == 0) {
    sum = warpSum(shared[tid]);
    if (tid == 0) {
      shared[0] = sum;
    }
  }
  __syncthreads();

  // Everyone picks it up, should be broadcast into the whole grad_input
  return shared[0];
}

template <typename scalar_t, typename accscalar_t, bool train, typename index_t>
__global__ void batch_norm_transform_input_kernel(
    const PackedTensorAccessor<scalar_t, 3, RestrictPtrTraits, index_t> input,
    PackedTensorAccessor<scalar_t, 3, RestrictPtrTraits, index_t> output,
    const PackedTensorAccessor<typename std::conditional<train, accscalar_t, scalar_t>::type, 1, RestrictPtrTraits, index_t> mean_,
    const PackedTensorAccessor<typename std::conditional<train, accscalar_t, scalar_t>::type, 1, RestrictPtrTraits, index_t> var_or_invstd,
    const PackedTensorAccessor<scalar_t, 1, RestrictPtrTraits, index_t> weight,
    const PackedTensorAccessor<scalar_t, 1, RestrictPtrTraits, index_t> bias,
    accscalar_t epsilon) {

  index_t plane = blockIdx.x;

  if (plane >= input.size(1)) {
    return;
  }

  accscalar_t gamma = weight.size(0) > 0 ? static_cast<accscalar_t>(weight[plane]) : static_cast<accscalar_t>(1);
  accscalar_t beta = bias.size(0) > 0 ? static_cast<accscalar_t>(bias[plane]) : static_cast<accscalar_t>(0);
  accscalar_t mean = static_cast<accscalar_t>(mean_[plane]);
  accscalar_t invstd;
  if (train) {
    invstd = var_or_invstd[plane];
  } else {
    invstd = static_cast<accscalar_t>(1) / device_sqrt(static_cast<accscalar_t>(var_or_invstd[plane]) + epsilon);
  }

  index_t bs = input.size(0);
  index_t fs = input.size(2);

  index_t bstep  = blockDim.y * gridDim.y;
  for (index_t batch = threadIdx.y + blockIdx.y * blockDim.y; batch < bs; batch += bstep) {
    auto o = output[batch][plane];
    auto i = input[batch][plane];
    for (index_t feature = threadIdx.x; feature < fs; feature += blockDim.x) {
      o[feature] = static_cast<scalar_t>(gamma * (i[feature] - mean) * invstd + beta);
    }
  }
}

template<typename T>
struct InvStd {
  __device__ __forceinline__ T operator()(T var, double epsilon) const {
    T invstd = 0;
    if (var != static_cast<T>(0) || epsilon != static_cast<T>(0)) {
      invstd = static_cast<T>(1) / device_sqrt(var + epsilon);
    }
    return invstd;
  }
};

template<typename T>
struct Var {
  __device__ __forceinline__ T operator()(T var, double epsilon) const {
    return var;
  }
};



template <template<typename T> class VarTransform0, typename input_scalar_t1, typename stat_scalar_t2, typename stat_accscalar_t3, typename index_t4>
__global__ void batch_norm_collect_statistics_kernel(
    const PackedTensorAccessor<input_scalar_t1, 3, RestrictPtrTraits, index_t4> input5,
    const stat_accscalar_t3 epsilon6,
    const stat_accscalar_t3 momentum7,
    PackedTensorAccessor<stat_scalar_t2, 1, RestrictPtrTraits, index_t4> running_mean8,
    PackedTensorAccessor<stat_scalar_t2, 1, RestrictPtrTraits, index_t4> running_var9,
    PackedTensorAccessor<stat_accscalar_t3, 1, RestrictPtrTraits, index_t4> save_mean10,
    PackedTensorAccessor<stat_accscalar_t3, 1, RestrictPtrTraits, index_t4> save_transformed_var11) {
    unsigned int blockDim_x_0 = 32;
    unsigned int threadIdx_x_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 32;
    unsigned int blockDim_y_0 = 16;
    unsigned int threadIdx_y_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 32 % 16;
    unsigned int blockDim_z_0 = 1;
    unsigned int threadIdx_z_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512;
    static int shared_n12[160] __attribute__((shared));
    int plane13 = blockIdx.x;
    int N14 = input5.size(0) * input5.size(2);
    int tid15 = threadIdx_x_0 + threadIdx_y_0 * blockDim_x_0;
    stat_accscalar_t3 *shared_avg_var16 = (stat_accscalar_t3 *)&shared_n12[WARP_SIZE];
    stat_accscalar_t3 avg17 = 0;
    stat_accscalar_t3 var_n18 = 0;
    int n19 = 0;
    for (int batch = threadIdx_y_0; batch < input5.size(0); batch += blockDim_y_0) {
        for (int x = threadIdx_x_0; x < input5.size(2); x += blockDim_x_0) {
            stat_accscalar_t3 v20 = input5[batch][plane13][x];
            stat_accscalar_t3 d121 = v20 - avg17;
            n19++;
            avg17 += d121 / n19;
            var_n18 += d121 * (v20 - avg17);
        }
    }
    for (int i = 0; i < getMSB(WARP_SIZE); ++i) {
        stat_accscalar_t3 o_avg22 = WARP_SHFL_XOR(avg17, 1 << i, WARP_SIZE);
        int o_n23 = WARP_SHFL_XOR(n19, 1 << i, WARP_SIZE);
        stat_accscalar_t3 factor24 = 1. / fmaxf(1., n19 + o_n23);
        var_n18 += WARP_SHFL_XOR(var_n18, 1 << i, WARP_SIZE) + (avg17 - o_avg22) * (avg17 - o_avg22) * n19 * o_n23 * factor24;
        avg17 = (n19 * avg17 + o_n23 * o_avg22) * factor24;
        n19 += o_n23;
    }
    __syncthreads();
    if (tid15 % WARP_SIZE == 0) {
        shared_n12[tid15 / WARP_SIZE] = n19;
        shared_avg_var16[tid15 / WARP_SIZE * 2] = avg17;
        shared_avg_var16[tid15 / WARP_SIZE * 2 + 1] = var_n18;
    }
    __syncthreads();
    if (tid15 < WARP_SIZE) {
        n19 = (tid15 < blockDim_x_0 * blockDim_y_0 / WARP_SIZE ? shared_n12[tid15] : 0);
        avg17 = (tid15 < blockDim_x_0 * blockDim_y_0 / WARP_SIZE ? shared_avg_var16[2 * tid15] : stat_accscalar_t3(0));
        var_n18 = (tid15 < blockDim_x_0 * blockDim_y_0 / WARP_SIZE ? shared_avg_var16[2 * tid15 + 1] : stat_accscalar_t3(0));
    }
    for (int i = 0; i < getMSB(WARP_SIZE); ++i) {
        stat_accscalar_t3 o_avg25 = WARP_SHFL_XOR(avg17, 1 << i, WARP_SIZE);
        int o_n26 = WARP_SHFL_XOR(n19, 1 << i, WARP_SIZE);
        stat_accscalar_t3 factor27 = 1. / fmaxf(1., n19 + o_n26);
        var_n18 += WARP_SHFL_XOR(var_n18, 1 << i, WARP_SIZE) + (avg17 - o_avg25) * (avg17 - o_avg25) * n19 * o_n26 * factor27;
        avg17 = (n19 * avg17 + o_n26 * o_avg25) * factor27;
        n19 += o_n26;
    }
    if (tid15 == 0) {
        if (save_mean10.data() != __null) {
            save_mean10[plane13] = avg17;
        }
        if (save_transformed_var11.data() != __null) {
            save_transformed_var11[plane13] = VarTransform0<stat_accscalar_t3>({})(var_n18 / N14, epsilon6);
        }
        if (running_mean8.data() != __null) {
            running_mean8[plane13] = static_cast<stat_scalar_t2>((1 - momentum7) * running_mean8[plane13] + momentum7 * avg17);
        }
        if (running_var9.data() != __null) {
            stat_accscalar_t3 unbiasedVar28 = var_n18 / (N14 - 1);
            running_var9[plane13] = static_cast<stat_scalar_t2>((1 - momentum7) * running_var9[plane13] + momentum7 * unbiasedVar28);
        }
    }
}




#define THRESH_NUMBER_BINS_FOR_MULTI_BLOCK_MEM 100
#define THRESH_NUMBER_BINS_FOR_GLOBAL_MEM 1000
#define FOR_KERNEL_LOOP(i, lim)                                      \
  for (IndexType i = blockIdx.x * blockDim.x + threadIdx.x; i < lim; \
       i += gridDim.x * blockDim.x)

/*
  Memory types used for the 3 histogram implementations.
  See `CUDA_tensor_histogram` below.
 */
enum class CUDAHistogramMemoryType { SHARED, MULTI_BLOCK, GLOBAL };
namespace {
  template<typename input_t, typename IndexType>
  __device__ static IndexType getBin(input_t bVal, input_t minvalue, input_t maxvalue, int nbins) {
    IndexType bin = (int)((bVal - minvalue) * nbins / (maxvalue - minvalue));
    // (only applicable for histc)
    // while each bin is inclusive at the lower end and exclusive at the higher, i.e. [start, end)
    // the last bin is inclusive at both, i.e. [start, end], in order to include maxvalue if exists
    // therefore when bin == nbins, adjust bin to the last bin
    if (bin == nbins) bin -= 1;
    return bin;
  }
}

/*
  Kernel for computing the histogram of the input.
 */
template <
    typename output_t29,
    typename input_t30,
    typename IndexType31,
    int ADims32,
    int PDims33,
    int BDims34,
    CUDAHistogramMemoryType MemoryType35 = CUDAHistogramMemoryType::MULTI_BLOCK,
    typename Op36>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(512)
#endif
__global__ void kernelHistogram1D(
    TensorInfo<output_t29, IndexType31> a37, /* output */
    TensorInfo<output_t29, IndexType31> p38, /* partial output */
    TensorInfo<input_t30, IndexType31> b39, /* input */
    int nbins40,
    input_t30 minvalue41,
    input_t30 maxvalue42,
    IndexType31 totalElements43,
    Op36 getOp44) {
    unsigned int blockDim_x_1 = 512;
    unsigned int threadIdx_x_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 512;
    unsigned int blockDim_y_1 = 1;
    unsigned int threadIdx_y_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512 % 1;
    unsigned int blockDim_z_1 = 1;
    unsigned int threadIdx_z_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512;
    extern unsigned char my_smem45[] __attribute__((shared));
    output_t29 *smem46 = nullptr;
    smem46 = reinterpret_cast<output_t29 *>(my_smem45);
    for (IndexType31 i = threadIdx_x_1; i < a37.sizes[0]; i += blockDim_x_1) {
        smem46[i] = 0;
    }
    __syncthreads();
    for (IndexType31 linearIndex = blockIdx.x * blockDim_x_1 + threadIdx_x_1; linearIndex < totalElements43; linearIndex += gridDim.x * blockDim_x_1) {
        const IndexType31 bOffset47 = IndexToOffset<input_t30, IndexType31, BDims34>::get(linearIndex, b39);
        const input_t30 bVal48 = b39.data[bOffset47];
        if (bVal48 >= minvalue41 && bVal48 <= maxvalue42) {
            const IndexType31 bin49 = getBin<input_t30, IndexType31>(bVal48, minvalue41, maxvalue42, nbins40);
            atomicAdd(& smem46[bin49], getOp44(linearIndex));
        }
    }
    __syncthreads();
    for (IndexType31 i = threadIdx_x_1; i < a37.sizes[0]; i += blockDim_x_1) {
        const IndexType31 aOffset50 = IndexToOffset<output_t29, IndexType31, ADims32>::get(i, a37);
        atomicAdd(& a37.data[aOffset50], smem46[i]);
    }
}



#include "kernelHistogram1D_batch_norm_collect_statistics_kernel_.inc"

inline int64_t getFreeGlobalMemory() {
  // no need to use `hipSetDevice`
  size_t free_mem, total_mem;
  hipMemGetInfo(&free_mem, &total_mem);
  AT_ASSERTM(
      hipGetLastError() == hipSuccess,
      "CUDA_tensor_histogram failed to get free global memory");
  return static_cast<int64_t>(free_mem);
}
template <typename input_hist_t, typename scalar_t, typename index_t>
std::tuple<Tensor, Tensor> _histc_cuda_template(
    const Tensor& self_hist,
    int64_t nbins,
    input_hist_t min,
    input_hist_t max,
    const Tensor& input_, double epsilon
    ) {
  printf("2\n");
  if (nbins <= 0) {
    AT_ERROR("bins must be > 0");
  }
  Tensor output_hist = native::zeros({nbins}, device(DeviceType::CUDA).dtype(self_hist.scalar_type()));
  input_hist_t minvalue = min;
  input_hist_t maxvalue = max;
  if (min == max) {
    minvalue = *self_hist.min().cpu().data<input_hist_t>();
    maxvalue = *self_hist.max().cpu().data<input_hist_t>();
  }
  if (minvalue == maxvalue) {
    minvalue = minvalue - 1;
    maxvalue = maxvalue + 1;
  }

  printf("3\n");
  {
  checkBackend("CUDA_tensor_histogram", {output_hist, self_hist}, Backend::CUDA);
  auto totalElements = self_hist.numel();

  const dim3 block = getApplyBlock();
  dim3 grid;
  int64_t curDevice = current_device();

  grid.x = 10000;

  CUDAHistogramMemoryType memType = CUDAHistogramMemoryType::GLOBAL;
  auto maxSharedMem = getCurrentDeviceProperties()->sharedMemPerBlock;
  auto sharedMem = nbins * sizeof(input_hist_t) + 8; // 8 guard bytes
  auto maxGlobalMem = getFreeGlobalMemory();
  auto multiBlockMem = nbins * grid.x * sizeof(input_hist_t) + 8; // 8 guard bytes
  // determine memory type to use in the kernel
    printf("6\n");
  if (nbins < THRESH_NUMBER_BINS_FOR_MULTI_BLOCK_MEM &&
      sharedMem < maxSharedMem) {
    printf("shared\n");
    memType = CUDAHistogramMemoryType::SHARED;
  } else if (
      nbins < THRESH_NUMBER_BINS_FOR_GLOBAL_MEM &&
      multiBlockMem < (maxGlobalMem / 2)) {
    // check against half of free mem to be extra safe
    // due to cached allocator, we may anyway have slightly more free mem
    printf("mb\n");
    memType = CUDAHistogramMemoryType::MULTI_BLOCK;
  }

  // alloc memory for MULTI_BLOCK
  using IndexType = int64_t;
  auto aInfo = getTensorInfo<input_hist_t, IndexType>(output_hist);
  auto bInfo = getTensorInfo<input_hist_t, IndexType>(self_hist);
  TensorInfo<input_hist_t, IndexType> pInfo(nullptr, 0, {}, {});
  Tensor partial_output_hist;
  if (memType == CUDAHistogramMemoryType::MULTI_BLOCK) {
    partial_output_hist = native::zeros({grid.x, nbins}, output_hist.options());
    pInfo = getTensorInfo<input_hist_t, IndexType>(partial_output_hist);
  }

  printf("7\n");
  printf("10\n");
  // Launch kernel
  using accscalar_t = at::acc_type<scalar_t, true>;
  int64_t n_input = input_.size(1);
  Tensor dummy_mean_;
  Tensor dummy_var_;
  Tensor mean_;
  Tensor invstd_;
  auto input_reshaped = input_.reshape({input_.size(0), input_.size(1), -1}); // internally we merge the feature dimensions

  auto bs = input_reshaped.size(0);
  auto features = input_reshaped.size(2);
  auto input = input_reshaped.packed_accessor<scalar_t, 3, RestrictPtrTraits, index_t>();
  auto input_options = input_.options();
  dummy_mean_ = at::empty({0}, input_options);
  dummy_var_ = at::empty({0}, input_options);
  // promote only mean_/invstd_ precision
  if (input_.scalar_type() == at::ScalarType::Half) {
    input_options = input_options.dtype(ScalarType::Float);
  }
  mean_ = at::empty({n_input}, input_options);
  invstd_ = at::empty({n_input}, input_options);
  auto mean = packed_accessor_or_dummy<accscalar_t, 1, RestrictPtrTraits, index_t>(mean_);
  auto invstd = packed_accessor_or_dummy<accscalar_t, 1, RestrictPtrTraits, index_t>(invstd_);
  auto dummy_mean = dummy_mean_.packed_accessor<scalar_t, 1, RestrictPtrTraits, index_t>();
  auto dummy_invstd = dummy_var_.packed_accessor<scalar_t, 1, RestrictPtrTraits, index_t>();
  auto stream = at::cuda::getCurrentCUDAStream();

  dim3 blocks(input.size(1));
  int tf = getNumThreads(input.size(2));
  dim3 threads(tf, std::max<int>(1, MAX_BLOCK_SIZE/tf));
  printf("input: %d %d \n", input.size(0), input.size(1));
  printf("%d %d %d\n", blocks.x, blocks.y, blocks.z);
  static const auto getDummyOp = [] __device__(IndexType) { return 1L; };
  hipProfilerStart();
  batch_norm_collect_statistics_kernel<InvStd, scalar_t, scalar_t, accscalar_t, index_t> <<<blocks, threads, 0, getStreamFromPool(true)>>>
    (input, epsilon, 0.0, dummy_mean, dummy_invstd, mean, invstd);
  kernelHistogram1D<input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
      <<<grid,
        block,
        sharedMem,
        getStreamFromPool(true)>>>(
          aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
  hipDeviceSynchronize();
  hipProfilerStop();
  AT_ASSERTM(hipGetLastError() == hipSuccess, "kernelHistogram1D failed");
  return std::make_tuple(output_hist, mean_);
}
}
template <typename input_hist_t, typename scalar_t, typename index_t>
std::tuple<Tensor, Tensor> _histc_cuda_fused(
    const Tensor& self_hist,
    int64_t nbins,
    input_hist_t min,
    input_hist_t max,
    const Tensor& input_, double epsilon
    ) {
  printf("2\n");
  if (nbins <= 0) {
    AT_ERROR("bins must be > 0");
  }
  Tensor output_hist = native::zeros({nbins}, device(DeviceType::CUDA).dtype(self_hist.scalar_type()));
  input_hist_t minvalue = min;
  input_hist_t maxvalue = max;
  if (min == max) {
    minvalue = *self_hist.min().cpu().data<input_hist_t>();
    maxvalue = *self_hist.max().cpu().data<input_hist_t>();
  }
  if (minvalue == maxvalue) {
    minvalue = minvalue - 1;
    maxvalue = maxvalue + 1;
  }

  printf("3\n");
  {
  checkBackend("CUDA_tensor_histogram", {output_hist, self_hist}, Backend::CUDA);
  auto totalElements = self_hist.numel();

  const dim3 block = getApplyBlock();
  dim3 grid;
  int64_t curDevice = current_device();

  grid.x = 10000;

  CUDAHistogramMemoryType memType = CUDAHistogramMemoryType::GLOBAL;
  auto maxSharedMem = getCurrentDeviceProperties()->sharedMemPerBlock;
  auto sharedMem = nbins * sizeof(input_hist_t) + 8; // 8 guard bytes
  auto maxGlobalMem = getFreeGlobalMemory();
  auto multiBlockMem = nbins * grid.x * sizeof(input_hist_t) + 8; // 8 guard bytes
  // determine memory type to use in the kernel
    printf("6\n");
  if (nbins < THRESH_NUMBER_BINS_FOR_MULTI_BLOCK_MEM &&
      sharedMem < maxSharedMem) {
    printf("shared\n");
    memType = CUDAHistogramMemoryType::SHARED;
  } else if (
      nbins < THRESH_NUMBER_BINS_FOR_GLOBAL_MEM &&
      multiBlockMem < (maxGlobalMem / 2)) {
    // check against half of free mem to be extra safe
    // due to cached allocator, we may anyway have slightly more free mem
    printf("mb\n");
    memType = CUDAHistogramMemoryType::MULTI_BLOCK;
  }

  // alloc memory for MULTI_BLOCK
  using IndexType = int64_t;
  auto aInfo = getTensorInfo<input_hist_t, IndexType>(output_hist);
  auto bInfo = getTensorInfo<input_hist_t, IndexType>(self_hist);
  TensorInfo<input_hist_t, IndexType> pInfo(nullptr, 0, {}, {});
  Tensor partial_output_hist;
  if (memType == CUDAHistogramMemoryType::MULTI_BLOCK) {
    partial_output_hist = native::zeros({grid.x, nbins}, output_hist.options());
    pInfo = getTensorInfo<input_hist_t, IndexType>(partial_output_hist);
  }

  printf("7\n");
  printf("10\n");
  // Launch kernel
  using accscalar_t = at::acc_type<scalar_t, true>;
  int64_t n_input = input_.size(1);
  Tensor dummy_mean_;
  Tensor dummy_var_;
  Tensor mean_;
  Tensor invstd_;
  auto input_reshaped = input_.reshape({input_.size(0), input_.size(1), -1}); // internally we merge the feature dimensions

  auto bs = input_reshaped.size(0);
  auto features = input_reshaped.size(2);
  auto input = input_reshaped.packed_accessor<scalar_t, 3, RestrictPtrTraits, index_t>();
  auto input_options = input_.options();
  dummy_mean_ = at::empty({0}, input_options);
  dummy_var_ = at::empty({0}, input_options);
  // promote only mean_/invstd_ precision
  if (input_.scalar_type() == at::ScalarType::Half) {
    input_options = input_options.dtype(ScalarType::Float);
  }
  mean_ = at::empty({n_input}, input_options);
  invstd_ = at::empty({n_input}, input_options);
  auto mean = packed_accessor_or_dummy<accscalar_t, 1, RestrictPtrTraits, index_t>(mean_);
  auto invstd = packed_accessor_or_dummy<accscalar_t, 1, RestrictPtrTraits, index_t>(invstd_);
  auto dummy_mean = dummy_mean_.packed_accessor<scalar_t, 1, RestrictPtrTraits, index_t>();
  auto dummy_invstd = dummy_var_.packed_accessor<scalar_t, 1, RestrictPtrTraits, index_t>();
  auto stream = at::cuda::getCurrentCUDAStream();

  dim3 blocks(input.size(1));
  int tf = getNumThreads(input.size(2));
  dim3 threads(tf, std::max<int>(1, MAX_BLOCK_SIZE/tf));
  printf("%d %d %d\n", blocks.x, blocks.y, blocks.z);
  THCudaCheck(hipGetLastError());

    static const auto getDummyOp = [] __device__(IndexType) { return 1L; };
    hipProfilerStart();
    #define CALL(i,type,thread) kernelHistogram1D_batch_norm_collect_statistics_kernel_fused_kernel_##type##_idx_##i<input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED, decltype(getDummyOp),\
  InvStd, scalar_t, scalar_t, accscalar_t, index_t> <<<10000, thread, sharedMem, stream>>>\
         (aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp,\
    input, epsilon, 0.0, dummy_mean, dummy_invstd, mean, invstd);\
    hipDeviceSynchronize()

      CALL(0, vfuse,512);
      CALL(0, vfuse_lb,512);
      CALL(0, hfuse,1024);
      CALL(0, hfuse_lb,1024);
      CALL(1, hfuse,1024);
      CALL(1, hfuse_lb,1024);
      CALL(2, hfuse,1024);
      CALL(2, hfuse_lb,1024);
      CALL(3, hfuse,1024);
      CALL(3, hfuse_lb,1024);
      CALL(4, hfuse,1024);
      CALL(4, hfuse_lb,1024);
      CALL(5, hfuse,1024);
      CALL(5, hfuse_lb,1024);
      CALL(6, hfuse,1024);
      CALL(6, hfuse_lb,1024);

  hipDeviceSynchronize();
    hipProfilerStop();
    AT_ASSERTM(hipGetLastError() == hipSuccess, "kernelHistogram1D failed");
  return std::make_tuple(output_hist, mean_);
}
}
} // namespace

namespace native {

std::tuple<Tensor, Tensor> hist_norm(
    const Tensor& self,
    int64_t nbins,
    Scalar min,
    Scalar max,
  Tensor& input_) {
  if (self.scalar_type() == ScalarType::Half) {
    AT_ERROR("HalfTensor is not supported");
  }
    printf("0\n");
  AT_DISPATCH_ALL_TYPES(self.scalar_type(), "histc", [&] {
    printf("1\n");
    return native::_histc_cuda_fused<scalar_t, scalar_t, int32_t>(self, nbins, min.to<scalar_t>(), max.to<scalar_t>()
    , input_, 0.2
  );
  });
  return AT_DISPATCH_ALL_TYPES(self.scalar_type(), "histc", [&] {
    printf("1\n");
    return native::_histc_cuda_template<scalar_t, scalar_t, int32_t>(self, nbins, min.to<scalar_t>(), max.to<scalar_t>()
    , input_, 0.2
  );
  });
}

} // namespace native
} // namespace at
