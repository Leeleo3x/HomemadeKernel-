#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <THC/THCGeneral.h>
#include <THC/THCDeviceUtils.cuh>

#include <ATen/ATen.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/div_rtn.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <c10/macros/Macros.h>
#include <ATen/native/im2col_shape_check.h>

#include <hip/hip_runtime_api.h>
namespace at {
namespace native {


using namespace at::cuda;
using namespace at::cuda::detail;

#include "im2col.inc"

#define THRESH_NUMBER_BINS_FOR_MULTI_BLOCK_MEM 100
#define THRESH_NUMBER_BINS_FOR_GLOBAL_MEM 1000
#define FOR_KERNEL_LOOP(i, lim)                                      \
  for (IndexType i = blockIdx.x * blockDim.x + threadIdx.x; i < lim; \
       i += gridDim.x * blockDim.x)

/*
  Memory types used for the 3 histogram implementations.
  See `CUDA_tensor_histogram` below.
 */
enum class CUDAHistogramMemoryType { SHARED, MULTI_BLOCK, GLOBAL };
namespace {
  template<typename input_t, typename IndexType>
  __device__ static IndexType getBin(input_t bVal, input_t minvalue, input_t maxvalue, int nbins) {
    IndexType bin = (int)((bVal - minvalue) * nbins / (maxvalue - minvalue));
    // (only applicable for histc)
    // while each bin is inclusive at the lower end and exclusive at the higher, i.e. [start, end)
    // the last bin is inclusive at both, i.e. [start, end], in order to include maxvalue if exists
    // therefore when bin == nbins, adjust bin to the last bin
    if (bin == nbins) bin -= 1;
    return bin;
  }
}

/*
  Kernel for computing the histogram of the input.
 */
template <
    typename output_t,
    typename input_t,
    typename IndexType,
    int ADims,
    int PDims,
    int BDims,
    CUDAHistogramMemoryType MemoryType = CUDAHistogramMemoryType::MULTI_BLOCK,
    typename Op>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(512)
#endif
__global__ void kernelHistogram1D(
    TensorInfo<output_t, IndexType> a, /* output */
    TensorInfo<output_t, IndexType> p, /* partial output */
    TensorInfo<input_t, IndexType> b, /* input */
    int nbins,
    input_t minvalue,
    input_t maxvalue,
    IndexType totalElements,
    Op getOp) {
  extern __shared__ unsigned char my_smem[];
  output_t* smem = nullptr;

    ////////////////////////// Shared memory //////////////////////////
    // atomically add to block specific shared memory
    // then atomically add to the global output tensor
    smem = reinterpret_cast<output_t*>(my_smem);
    for (IndexType i = threadIdx.x; i < a.sizes[0]; i += blockDim.x) {
      smem[i] = 0;
    }
    __syncthreads();
    FOR_KERNEL_LOOP(linearIndex, totalElements) {
      // Convert `linearIndex` into an offset of `b`
      const IndexType bOffset =
          IndexToOffset<input_t, IndexType, BDims>::get(linearIndex, b);
      const input_t bVal = b.data[bOffset];
      if (bVal >= minvalue && bVal <= maxvalue) {
        // Use value at `b` as an offset of `smem`
        const IndexType bin = getBin<input_t, IndexType>(bVal, minvalue, maxvalue, nbins);
        atomicAdd(&smem[bin], getOp(linearIndex));
      }
    }
    __syncthreads();
    // NOTE: atomically update output bin count.
    //   Atomic update is imp since __syncthread() will only synchronize threads
    //   in a given block, not across blocks.
    for (IndexType i = threadIdx.x; i < a.sizes[0]; i += blockDim.x) {
      const IndexType aOffset =
          IndexToOffset<output_t, IndexType, ADims>::get(i, a);
      atomicAdd(&a.data[aOffset], smem[i]);
    }

}

#include "im2col_kernel_kernelHistogram1D_.inc"

inline int64_t getFreeGlobalMemory() {
  // no need to use `hipSetDevice`
  size_t free_mem, total_mem;
  hipMemGetInfo(&free_mem, &total_mem);
  AT_ASSERTM(
      hipGetLastError() == hipSuccess,
      "CUDA_tensor_histogram failed to get free global memory");
  return static_cast<int64_t>(free_mem);
}

template <typename input_hist_t>
std::tuple<Tensor, Tensor> _histc_cuda_template_fused(
    const Tensor& self_hist,
    int64_t nbins,
    input_hist_t min,
    input_hist_t max,
    const Tensor& input_,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride) {
  printf("2\n");
  if (nbins <= 0) {
    AT_ERROR("bins must be > 0");
  }
  Tensor output_hist = native::zeros({nbins}, device(DeviceType::CUDA).dtype(self_hist.scalar_type()));
  input_hist_t minvalue = min;
  input_hist_t maxvalue = max;
  if (min == max) {
    minvalue = *self_hist.min().cpu().data<input_hist_t>();
    maxvalue = *self_hist.max().cpu().data<input_hist_t>();
  }
  if (minvalue == maxvalue) {
    minvalue = minvalue - 1;
    maxvalue = maxvalue + 1;
  }

  printf("3\n");
  {
  checkBackend("CUDA_tensor_histogram", {output_hist, self_hist}, Backend::CUDA);
  auto totalElements = self_hist.numel();

  const dim3 block = getApplyBlock();
  dim3 grid;
  int64_t curDevice = current_device();

  grid.x = 10000;

  CUDAHistogramMemoryType memType = CUDAHistogramMemoryType::GLOBAL;
  auto maxSharedMem = getCurrentDeviceProperties()->sharedMemPerBlock;
  auto sharedMem = nbins * sizeof(input_hist_t) + 8; // 8 guard bytes
  auto maxGlobalMem = getFreeGlobalMemory();
  auto multiBlockMem = nbins * grid.x * sizeof(input_hist_t) + 8; // 8 guard bytes
  // determine memory type to use in the kernel
    printf("6\n");
  if (nbins < THRESH_NUMBER_BINS_FOR_MULTI_BLOCK_MEM &&
      sharedMem < maxSharedMem) {
    printf("shared\n");
    memType = CUDAHistogramMemoryType::SHARED;
  } else if (
      nbins < THRESH_NUMBER_BINS_FOR_GLOBAL_MEM &&
      multiBlockMem < (maxGlobalMem / 2)) {
    // check against half of free mem to be extra safe
    // due to cached allocator, we may anyway have slightly more free mem
    printf("mb\n");
    memType = CUDAHistogramMemoryType::MULTI_BLOCK;
  }

  // alloc memory for MULTI_BLOCK
  using IndexType = int64_t;
  auto aInfo = getTensorInfo<input_hist_t, IndexType>(output_hist);
  auto bInfo = getTensorInfo<input_hist_t, IndexType>(self_hist);
  TensorInfo<input_hist_t, IndexType> pInfo(nullptr, 0, {}, {});
  Tensor partial_output_hist;
  if (memType == CUDAHistogramMemoryType::MULTI_BLOCK) {
    partial_output_hist = native::zeros({grid.x, nbins}, output_hist.options());
    pInfo = getTensorInfo<input_hist_t, IndexType>(partial_output_hist);
  }

  printf("7\n");
  printf("10\n");
  Tensor output = at::empty_like(input_);
  int64_t kernel_height = kernel_size[0];
  int64_t kernel_width = kernel_size[1];
  int64_t dilation_height = dilation[0];
  int64_t dilation_width = dilation[1];
  int64_t pad_height = padding[0];
  int64_t pad_width = padding[1];
  int64_t stride_height = stride[0];
  int64_t stride_width = stride[1];

  TensorArg input_arg{input_, "input", 1};
  TensorArg output_arg{output, "output", 2};
  checkAllSameGPU("im2col_cuda", {input_arg, output_arg});

  im2col_shape_check(
      input_,
      Tensor(),
      kernel_height,
      kernel_width,
      dilation_height,
      dilation_width,
      pad_height,
      pad_width,
      stride_height,
      stride_width);

  Tensor input = input_.contiguous();

  bool batched_input = true;

  if (input.dim() == 3) {
    batched_input = false;
    input.resize_({1, input.size(0), input.size(1), input.size(2)});
  }

  int64_t batch_size = input.size(0);
  int64_t n_input_plane = input.size(1);
  int64_t input_height = input.size(2);
  int64_t input_width = input.size(3);

  int64_t output_height = (input_height + 2 * pad_height -
                           (dilation_height * (kernel_height - 1) + 1)) /
          stride_height +
      1;
  int64_t output_width = (input_width + 2 * pad_width -
                          (dilation_width * (kernel_width - 1) + 1)) /
          stride_width +
      1;
  int64_t n_output_plane = n_input_plane * kernel_width * kernel_height;
  int64_t output_length = output_height * output_width;

  output.resize_({batch_size, n_output_plane, output_length});
  output.zero_();

  // Launch kernel
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "im2col_out_cuda", [&] {
    Tensor input_n;
    Tensor output_n;

    int64_t elt = 0;
    input_n = input.select(0, elt);
    output_n = output.select(0, elt);
    int64_t num_kernels = n_input_plane * output_height * output_width;
    static const auto getDummyOp = [] __device__(IndexType) { return 1L; };

  hipProfilerStart();
    im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_bar_sync_idx_0
    <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels,
        input_n.data<scalar_t>(),
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output_height,
        output_width,
        output_n.data<scalar_t>(),
        aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_lb_bar_sync_idx_0
    <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels,
        input_n.data<scalar_t>(),
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output_height,
        output_width,
        output_n.data<scalar_t>(),
        aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_idx_1
    <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels,
        input_n.data<scalar_t>(),
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output_height,
        output_width,
        output_n.data<scalar_t>(),
        aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_lb_idx_1
    <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels,
        input_n.data<scalar_t>(),
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output_height,
        output_width,
        output_n.data<scalar_t>(),
        aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_idx_0
    <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels,
        input_n.data<scalar_t>(),
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output_height,
        output_width,
        output_n.data<scalar_t>(),
        aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_lb_idx_0
    <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels,
        input_n.data<scalar_t>(),
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output_height,
        output_width,
        output_n.data<scalar_t>(),
        aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    im2col_kernel_kernelHistogram1D_fused_kernel_vfuse_lb_idx_0
    <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    <<<10000, 512, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels,
        input_n.data<scalar_t>(),
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output_height,
        output_width,
        output_n.data<scalar_t>(),
        aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    im2col_kernel_kernelHistogram1D_fused_kernel_vfuse_idx_0
    <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    <<<10000, 512, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels,
        input_n.data<scalar_t>(),
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output_height,
        output_width,
        output_n.data<scalar_t>(),
        aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);

    im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_bar_sync_imba_idx_0
    <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels,
        input_n.data<scalar_t>(),
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output_height,
        output_width,
        output_n.data<scalar_t>(),
        aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_lb_bar_sync_imba_idx_0
    <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels,
        input_n.data<scalar_t>(),
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output_height,
        output_width,
        output_n.data<scalar_t>(),
        aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    // im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_imba_idx_1
    // <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    // <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
    //     num_kernels,
    //     input_n.data<scalar_t>(),
    //     input_height,
    //     input_width,
    //     kernel_height,
    //     kernel_width,
    //     pad_height,
    //     pad_width,
    //     stride_height,
    //     stride_width,
    //     dilation_height,
    //     dilation_width,
    //     output_height,
    //     output_width,
    //     output_n.data<scalar_t>(),
    //     aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    // im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_lb_imba_idx_1
    // <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    // <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
    //     num_kernels,
    //     input_n.data<scalar_t>(),
    //     input_height,
    //     input_width,
    //     kernel_height,
    //     kernel_width,
    //     pad_height,
    //     pad_width,
    //     stride_height,
    //     stride_width,
    //     dilation_height,
    //     dilation_width,
    //     output_height,
    //     output_width,
    //     output_n.data<scalar_t>(),
    //     aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    // im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_imba_idx_0
    // <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    // <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
    //     num_kernels,
    //     input_n.data<scalar_t>(),
    //     input_height,
    //     input_width,
    //     kernel_height,
    //     kernel_width,
    //     pad_height,
    //     pad_width,
    //     stride_height,
    //     stride_width,
    //     dilation_height,
    //     dilation_width,
    //     output_height,
    //     output_width,
    //     output_n.data<scalar_t>(),
    //     aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
    // im2col_kernel_kernelHistogram1D_fused_kernel_hfuse_lb_imba_idx_0
    // <scalar_t, input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
    // <<<10000, 1024, sharedMem, at::cuda::getCurrentCUDAStream()>>>(
    //     num_kernels,
    //     input_n.data<scalar_t>(),
    //     input_height,
    //     input_width,
    //     kernel_height,
    //     kernel_width,
    //     pad_height,
    //     pad_width,
    //     stride_height,
    //     stride_width,
    //     dilation_height,
    //     dilation_width,
    //     output_height,
    //     output_width,
    //     output_n.data<scalar_t>(),
    //     aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
  hipProfilerStop();
    AT_ASSERTM(hipGetLastError() == hipSuccess, "kernelHistogram1D failed");
    if (!batched_input) {
      output.resize_({n_output_plane, output_length});
    }
  });
  hipDeviceSynchronize();
  return std::make_tuple(output_hist, output);
}
}

template <typename input_hist_t>
std::tuple<Tensor, Tensor> _histc_cuda_template(
    const Tensor& self_hist,
    int64_t nbins,
    input_hist_t min,
    input_hist_t max,
    const Tensor& input_,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride) {
  printf("2\n");
  if (nbins <= 0) {
    AT_ERROR("bins must be > 0");
  }
  Tensor output_hist = native::zeros({nbins}, device(DeviceType::CUDA).dtype(self_hist.scalar_type()));
  input_hist_t minvalue = min;
  input_hist_t maxvalue = max;
  if (min == max) {
    minvalue = *self_hist.min().cpu().data<input_hist_t>();
    maxvalue = *self_hist.max().cpu().data<input_hist_t>();
  }
  if (minvalue == maxvalue) {
    minvalue = minvalue - 1;
    maxvalue = maxvalue + 1;
  }

  printf("3\n");
  {
  checkBackend("CUDA_tensor_histogram", {output_hist, self_hist}, Backend::CUDA);
  auto totalElements = self_hist.numel();

  const dim3 block = getApplyBlock();
  dim3 grid;
  int64_t curDevice = current_device();

  grid.x = 10000;

  CUDAHistogramMemoryType memType = CUDAHistogramMemoryType::GLOBAL;
  auto maxSharedMem = getCurrentDeviceProperties()->sharedMemPerBlock;
  auto sharedMem = nbins * sizeof(input_hist_t) + 8; // 8 guard bytes
  auto maxGlobalMem = getFreeGlobalMemory();
  auto multiBlockMem = nbins * grid.x * sizeof(input_hist_t) + 8; // 8 guard bytes
  // determine memory type to use in the kernel
    printf("6\n");
  if (nbins < THRESH_NUMBER_BINS_FOR_MULTI_BLOCK_MEM &&
      sharedMem < maxSharedMem) {
    printf("shared\n");
    memType = CUDAHistogramMemoryType::SHARED;
  } else if (
      nbins < THRESH_NUMBER_BINS_FOR_GLOBAL_MEM &&
      multiBlockMem < (maxGlobalMem / 2)) {
    // check against half of free mem to be extra safe
    // due to cached allocator, we may anyway have slightly more free mem
    printf("mb\n");
    memType = CUDAHistogramMemoryType::MULTI_BLOCK;
  }

  // alloc memory for MULTI_BLOCK
  using IndexType = int64_t;
  auto aInfo = getTensorInfo<input_hist_t, IndexType>(output_hist);
  auto bInfo = getTensorInfo<input_hist_t, IndexType>(self_hist);
  TensorInfo<input_hist_t, IndexType> pInfo(nullptr, 0, {}, {});
  Tensor partial_output_hist;
  if (memType == CUDAHistogramMemoryType::MULTI_BLOCK) {
    partial_output_hist = native::zeros({grid.x, nbins}, output_hist.options());
    pInfo = getTensorInfo<input_hist_t, IndexType>(partial_output_hist);
  }

  printf("7\n");
  printf("10\n");
  Tensor output = at::empty_like(input_);
  int64_t kernel_height = kernel_size[0];
  int64_t kernel_width = kernel_size[1];
  int64_t dilation_height = dilation[0];
  int64_t dilation_width = dilation[1];
  int64_t pad_height = padding[0];
  int64_t pad_width = padding[1];
  int64_t stride_height = stride[0];
  int64_t stride_width = stride[1];

  TensorArg input_arg{input_, "input", 1};
  TensorArg output_arg{output, "output", 2};
  checkAllSameGPU("im2col_cuda", {input_arg, output_arg});

  im2col_shape_check(
      input_,
      Tensor(),
      kernel_height,
      kernel_width,
      dilation_height,
      dilation_width,
      pad_height,
      pad_width,
      stride_height,
      stride_width);

  Tensor input = input_.contiguous();

  bool batched_input = true;

  if (input.dim() == 3) {
    batched_input = false;
    input.resize_({1, input.size(0), input.size(1), input.size(2)});
  }

  int64_t batch_size = input.size(0);
  int64_t n_input_plane = input.size(1);
  int64_t input_height = input.size(2);
  int64_t input_width = input.size(3);

  int64_t output_height = (input_height + 2 * pad_height -
                           (dilation_height * (kernel_height - 1) + 1)) /
          stride_height +
      1;
  int64_t output_width = (input_width + 2 * pad_width -
                          (dilation_width * (kernel_width - 1) + 1)) /
          stride_width +
      1;
  int64_t n_output_plane = n_input_plane * kernel_width * kernel_height;
  int64_t output_length = output_height * output_width;

  output.resize_({batch_size, n_output_plane, output_length});
  output.zero_();

  // Launch kernel
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "im2col_out_cuda", [&] {
    Tensor input_n;
    Tensor output_n;

    int64_t elt = 0;
    input_n = input.select(0, elt);
    output_n = output.select(0, elt);
    int64_t num_kernels = n_input_plane * output_height * output_width;
    hipProfilerStart();
    im2col_kernel<<<10000, 512, 0, at::cuda::getStreamFromPool(true)>>>(
        num_kernels,
        input_n.data<scalar_t>(),
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output_height,
        output_width,
        output_n.data<scalar_t>());

    static const auto getDummyOp = [] __device__(IndexType) { return 1L; };
    kernelHistogram1D<input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
        <<<grid,
          block,
          sharedMem,
          getStreamFromPool(true)>>>(
            aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);        \
    hipProfilerStop();
    hipDeviceSynchronize();
    AT_ASSERTM(hipGetLastError() == hipSuccess, "kernelHistogram1D failed");
    if (!batched_input) {
      output.resize_({n_output_plane, output_length});
    }
  });
  return std::make_tuple(output_hist, output);
}
}
} // namespace

namespace native {

std::tuple<Tensor, Tensor> _histc_cuda2(
  const Tensor& input_im2col_,
  IntArrayRef kernel_size_im2col,
  IntArrayRef dilation_im2col,
  IntArrayRef pad_im2colding_im2col,
  IntArrayRef stride_im2col,
    const Tensor& self,
    int64_t nbins,
    Scalar min,
    Scalar max) {
  if (self.scalar_type() == ScalarType::Half) {
    AT_ERROR("HalfTensor is not supported");
  }
    printf("0\n");
  AT_DISPATCH_ALL_TYPES(self.scalar_type(), "histc", [&] {
    printf("1\n");
    return native::_histc_cuda_template<scalar_t>(self, nbins, min.to<scalar_t>(), max.to<scalar_t>(),
    input_im2col_,
    kernel_size_im2col,
    dilation_im2col,
    pad_im2colding_im2col,
    stride_im2col
  );
  });
  return AT_DISPATCH_ALL_TYPES(self.scalar_type(), "histc", [&] {
    printf("1\n");
    return native::_histc_cuda_template_fused<scalar_t>(self, nbins, min.to<scalar_t>(), max.to<scalar_t>(),
    input_im2col_,
    kernel_size_im2col,
    dilation_im2col,
    pad_im2colding_im2col,
    stride_im2col
  );
  });
}

} // namespace native
} // namespace at
